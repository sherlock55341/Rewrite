#include "aig.hpp"

RW_NAMESPACE_START

void AIGManager::resetManager() {
    designName_ = "";
    designPath_ = "";
    comments_ = "";

    nObjs = 0;
    nPIs = 0;
    nPOs = 0;
    nLatches = 0;
    nNodes = 0;
    nLevels = 0;

    if (pFanin0) {
        free(pFanin0);
    }
    pFanin0 = nullptr;
    if (pFanin1) {
        free(pFanin1);
    }
    pFanin1 = nullptr;
    if (pNumFanouts) {
        free(pNumFanouts);
    }
    pNumFanouts = nullptr;
    if (pOuts) {
        free(pOuts);
    }
    pOuts = nullptr;

    if(d_pFanin0){
        hipFree(d_pFanin0);
    }
    d_pFanin0 = nullptr;
    if(d_pFanin1){
        hipFree(d_pFanin1);
    }
    d_pFanin1 = nullptr;
    if(d_pNumFanouts){
        hipFree(d_pNumFanouts);
    }
    d_pNumFanouts = nullptr;
    if(d_pOuts){
        hipFree(d_pOuts);
    }
    d_pOuts = nullptr;

    aigCreated = false;
    
    aigOnDevice = false;  

    usingDar = false;

    aigNewest = true;
}

void AIGManager::mallocDevice(){
    if(aigOnDevice){
        printf("Error : AIG has already on device\n");
        return ;
    }
    aigOnDevice = true;
    hipMalloc(&d_pFanin0, sizeof(int) * nObjs);
    hipMalloc(&d_pFanin1, sizeof(int) * nObjs);
    hipMalloc(&d_pNumFanouts, sizeof(int) * nObjs);
    hipMalloc(&pOuts, sizeof(int) * nPOs);
}

void AIGManager::freeDevice(){
    if(!aigOnDevice){
        printf("Error : AIG has not been created on device\n");
        return ;
    }
    hipFree(d_pFanin0);
    hipFree(d_pFanin1);
    hipFree(d_pNumFanouts);
    hipFree(d_pOuts);
}

void AIGManager::copy2Device() const{
    if(!aigOnDevice){
        printf("Error : AIG has not been created on device\n");
        return ;
    }
    hipMemcpy(d_pFanin0, pFanin0, sizeof(int) * nObjs, hipMemcpyHostToDevice);
    hipMemcpy(d_pFanin1, pFanin1, sizeof(int) * nObjs, hipMemcpyHostToDevice);
    hipMemcpy(d_pNumFanouts, pNumFanouts, sizeof(int) * nObjs, hipMemcpyHostToDevice);
    hipMemcpy(d_pOuts, pOuts, sizeof(int) * nPOs, hipMemcpyHostToDevice);
}

void AIGManager::copyFromDevice() const{
    if(!aigOnDevice){
        printf("Error : AIG has not been created on device\n");
        return ;
    }
    hipMemcpy(pFanin0, d_pFanin0, sizeof(int) * nObjs, hipMemcpyDeviceToHost);
    hipMemcpy(pFanin1, d_pFanin1, sizeof(int) * nObjs, hipMemcpyDeviceToHost);
    hipMemcpy(pNumFanouts, d_pNumFanouts, sizeof(int) * nObjs, hipMemcpyDeviceToHost);
    hipMemcpy(pOuts, d_pOuts, sizeof(int) * nPOs, hipMemcpyDeviceToHost);
}

RW_NAMESPACE_END